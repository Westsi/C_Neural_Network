#include <hip/hip_runtime.h>
#include <stdio.h>

#define CUDA_CHECK(call) \
{ \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        fprintf(stderr, "CUDA Error: %s (error code %d) at %s:%d\n", \
                hipGetErrorString(err), err, __FILE__, __LINE__); \
        exit(EXIT_FAILURE); \
    } \
}

void checkDevices() {
    int deviceCount;
    CUDA_CHECK(hipGetDeviceCount(&deviceCount));
    printf("CUDA Device Count: %d\n", deviceCount);
}