#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

#include "nvcalls.h"

#define CUDA_CHECK(call) \
{ \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        fprintf(stderr, "CUDA Error: %s (error code %d) at %s:%d\n", \
                hipGetErrorString(err), err, __FILE__, __LINE__); \
        exit(EXIT_FAILURE); \
    } \
}

void checkDevices() {
    int deviceCount;
    CUDA_CHECK(hipGetDeviceCount(&deviceCount));
    printf("CUDA Device Count: %d\n", deviceCount);
}

__global__ void multiplyElements(const float *inputs, const float *weights, float *results, int numElements) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < numElements) {
        results[i] = inputs[i] * weights[i];
    }
}

__global__ void reduceSum(const float *input, float *output, int numElements) {
    extern __shared__ float sdata[];
    int tid = threadIdx.x;
    int i = blockDim.x * blockIdx.x + tid;

    sdata[tid] = (i < numElements) ? input[i] : 0.0f;
    __syncthreads();

    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    if (tid == 0) {
        output[blockIdx.x] = sdata[0];
    }
}

float computeDotProduct(const float *inputs, const float *weights, int numElements) {
    float *d_inputs, *d_weights, *d_results, *d_partialSums;
    int blockSize = 256;
    int numBlocks = (numElements + blockSize - 1) / blockSize;

    hipMalloc((void**)&d_inputs, numElements * sizeof(float));
    hipMalloc((void**)&d_weights, numElements * sizeof(float));
    hipMalloc((void**)&d_results, numElements * sizeof(float));
    hipMalloc((void**)&d_partialSums, numBlocks * sizeof(float));

    hipMemcpy(d_inputs, inputs, numElements * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_weights, weights, numElements * sizeof(float), hipMemcpyHostToDevice);

    multiplyElements<<<numBlocks, blockSize>>>(d_inputs, d_weights, d_results, numElements);
    reduceSum<<<numBlocks, blockSize, blockSize * sizeof(float)>>>(d_results, d_partialSums, numElements);

    float *h_partialSums = (float*)malloc(numBlocks * sizeof(float));
    hipMemcpy(h_partialSums, d_partialSums, numBlocks * sizeof(float), hipMemcpyDeviceToHost);

    float result = 0.0f;
    for (int i = 0; i < numBlocks; i++) {
        result += h_partialSums[i];
    }

    hipFree(d_inputs);
    hipFree(d_weights);
    hipFree(d_results);
    hipFree(d_partialSums);
    free(h_partialSums);
    return result;
}